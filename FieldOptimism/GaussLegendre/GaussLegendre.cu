#include "hip/hip_runtime.h"
#include "GaussLegendre.cuh"

void GaussLegendre::initNodeWeight(const int _M, const int _N)
{
	M = _M;
	N = _N;

	calcWeight(M, d_node_row, d_weight_row);
	calcWeight(N, d_node_col, d_weight_col);
}

__device__
float GaussLegendre::calcInte(const float4& x, const float4& y, const float sigma, const float ratio)
{
	
	float sum = 0.0;
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			float2 map_v = map(x, y, d_node_row[i], d_node_col[j]);
			sum += d_weight_row[i] * d_weight_col[j] * jacobi(x, y, d_node_row[i], d_node_col[j])*flux_func(map_v.x, map_v.y, sigma, ratio);
		}
	}
	return sum;
}


void GaussLegendre::legendre(const float t, const float m, float&p, float& dp)
{
	float p0 = 1.0;
	float p1 = t;
	for (int k = 1; k < m; k++) {
		p = ((2.0*k + 1)*t*p1 - k*p0) / (1.0 + k);
		p0 = p1;
		p1 = p;
	}
	dp = m*(p0 - t*p1) / (1.0 - t*t);
}


///
//��������������x1��x2������n�����س���Ϊn��x_list��w_list��
//���зֱ���n��gauss - legendre����ֹ�ʽ������㼰Ȩ��
//	:param x1 : ��������
//	:param x2 : ��������
//	:param x : ����ֹ�ʽ�������
//	:param w : ����ֹ�ʽ��Ȩ��
//	:param n : ��˹���ֽ���
void GaussLegendre::calcWeight(const int n, float*& x, float*& w, const float a, const float b) {
	float * h_node = new float[n];
	float * h_weight = new float[n];
	hipMalloc((void**)&x, sizeof(float)*n);
	hipMalloc((void**)&w, sizeof(float)*n);

	int nRoots = int((n + 1) / 2);
	float p, dp, dt, t;
	for (int i = 0; i < nRoots; i++) {
		t = cos(PI*(i + 0.75) / (n + 0.5));
		while (true) {
			legendre(t, n, p, dp);
			dt = -p / dp;
			t += dt;
			if (abs(dt) < Epsilon) {
				h_node[i] = -t;
				h_node[n - 1 - i] = t;
				h_weight[i] = 2.0 / (1.0 - t*t) / (dp*dp);
				h_weight[n - i - 1] = h_weight[i];
				break;
			}
		}
	}

	hipMemcpy(x, h_node, sizeof(float)*n, hipMemcpyHostToDevice);
	hipMemcpy(w, h_weight, sizeof(float)*n, hipMemcpyHostToDevice);

	delete[] h_node;
	delete[] h_weight;
}

__device__ 
float GaussLegendre::jacobi(const float4& x, const float4& y, const float s, const float t) {
	float J00 = -(1.0 - t) * x.x + (1.0 - t) * x.y + (1.0 + t) * x.z - (1.0 + t) * x.w;
	float J01 = -(1.0 - t) * y.x + (1.0 - t) * y.y + (1.0 + t) * y.z - (1.0 + t) * y.w;
	float J10 = -(1.0 - s) * x.x - (1.0 + s) * x.y + (1.0 + s) * x.z + (1.0 - s) * x.w;
	float J11 = -(1.0 - s) * y.x - (1.0 + s) * y.y + (1.0 + s) * y.z + (1.0 - s) * y.w;
	return (J00*J11 - J01*J10) / 16.0;
}


__device__
float2 GaussLegendre::map(const float4&x, const float4&y, const float s, const float t) {
	float4 N;
	N.x = (1.0 - s)*(1.0 - t) / 4.0;
	N.y = (1.0 + s)*(1.0 - t) / 4.0;
	N.z = (1.0 + s)*(1.0 + t) / 4.0;
	N.w = (1.0 - s)*(1.0 + t) / 4.0;
	float2 map_v;
	map_v.x = dot(N, x);
	map_v.y = dot(N, y);
	return map_v;
}

