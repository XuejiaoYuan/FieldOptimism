#include "HelioEnergy.cuh"


vector<float> HelioEnergy::calcHelioEnergy(FieldUpdateMode mode)
{
	int helioNum = solar_scene->helios.size();
	float* h_helio_energy = new float[helioNum];
	for (int i = 0; i < helioNum; ++i) h_helio_energy[i] = 0;

	int nThreads = 512;
	dim3 nBlocks;
	GeometryFunc::setThreadsBlocks(nBlocks, nThreads, r_args.numberOfReceivers*m*n*helioNum);

	switch (mode)
	{
	case HelioUpdateMode:
		h_args.setHelioDevicePos(solar_scene->helios);
	case SunUpdateMode:
		h_args.setHelioDeviceArguments(solar_scene->helios);
		h_args.setHelioRecvArguments(solar_scene->helios, *(solar_scene->recvs[0]));
		if(calcCenterMode) h_args.setHelioCenterBias(solar_scene->helios);
		break;
	default:
		break;
	}

	float* d_helio_energy = nullptr;
	hipMalloc((void**)&d_helio_energy, sizeof(float)*helioNum);
	hipMemcpy(d_helio_energy, h_helio_energy, sizeof(float)*helioNum, hipMemcpyHostToDevice);
	calcHelioFluxIntegral << <nBlocks, nThreads >> > (h_args, r_args, gl_handler, d_helio_energy, m, n);
	hipDeviceSynchronize();
	hipMemcpy(h_helio_energy, d_helio_energy, sizeof(float)*helioNum, hipMemcpyDeviceToHost);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
	}

	vector<float> res;
	for (int i = 0; i < helioNum; ++i) res.push_back(h_helio_energy[i]);

	delete[] h_helio_energy;
	h_helio_energy = nullptr;
	hipFree(d_helio_energy);
	return res;
}

float HelioEnergy::calcTotalEnergy(FieldUpdateMode mode)
{
	int helioNum = solar_scene->helios.size();
	float* h_total_energy = new float;
	*h_total_energy = 0;

	int nThreads = 512;
	dim3 nBlocks;
	GeometryFunc::setThreadsBlocks(nBlocks, nThreads, r_args.numberOfReceivers*m*n*helioNum);

	switch (mode)
	{
	case HelioUpdateMode:
		h_args.setHelioDevicePos(solar_scene->helios);
	case SunUpdateMode:
		h_args.setHelioDeviceArguments(solar_scene->helios);
		h_args.setHelioRecvArguments(solar_scene->helios, *(solar_scene->recvs[0]));
		if(calcCenterMode) h_args.setHelioCenterBias(solar_scene->helios);
		break;
	default:
		break;
	}

	hipMemcpy(d_total_energy, h_total_energy, sizeof(float), hipMemcpyHostToDevice);
	calcFieldFluxIntegral << <nBlocks, nThreads >> > (h_args, r_args, gl_handler, d_total_energy, m, n);
	hipDeviceSynchronize();
	hipMemcpy(h_total_energy, d_total_energy, sizeof(float), hipMemcpyDeviceToHost);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
	}

	float res = *h_total_energy;
	delete h_total_energy;
	h_total_energy = nullptr;
	return res;
}
