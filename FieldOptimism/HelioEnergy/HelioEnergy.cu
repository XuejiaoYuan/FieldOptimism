#include "HelioEnergy.cuh"


void HelioEnergy::calcHelioEnergy(float sigma, FieldUpdateMode mode)
{
	int helioNum = solar_scene->helios.size();
	float* h_total_energy = new float;
	*h_total_energy = 0;

	hipMemcpy(d_total_energy, h_total_energy, sizeof(float), hipMemcpyHostToDevice);

	int nThreads = 1024;
	dim3 nBlocks;
	GeometryFunc::setThreadsBlocks(nBlocks, nThreads, r_args.numberOfReceivers*m*n*helioNum);

	h_args.sigma = sigma;
	switch (mode)
	{
	case HelioUpdateMode:
		h_args.setHelioDevicePos(solar_scene->helios);
	case SunUpdateMode:
		h_args.setHelioDeviceArguments(solar_scene->helios);
		h_args.setHelioRecvArguments(solar_scene->helios, *(solar_scene->recvs[0]));
		break;
	default:
		break;
	}

	fluxIntegral << <nBlocks, nThreads >> > (h_args, r_args, gl_handler, d_total_energy, m, n);
	hipDeviceSynchronize();

	hipMemcpy(h_total_energy, d_total_energy, sizeof(float), hipMemcpyDeviceToHost);

	cout << *h_total_energy << endl;
	delete h_total_energy;
}
