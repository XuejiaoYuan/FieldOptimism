#include "hip/hip_runtime.h"
#include "FluxIntegral.cuh"

__global__ void fluxIntegral(IntegralHelioDeviceArgumet h_args, ReceiverDeviceArgument r_args, GaussLegendre gl, float* d_total_energy, const int m, const int n) {
	int myId = GeometryFunc::getThreadId();
	if (myId >= m*n*h_args.numberOfHeliostats*r_args.numberOfReceivers) return;

	int helioIndex = myId / (m*n*r_args.numberOfReceivers);
	int recvIndex = (myId % (m*n*r_args.numberOfReceivers)) / (m*n);
	int row_col = (myId % (m*n*r_args.numberOfReceivers)) % (m*n);
	int i = row_col / n;
	int j = row_col % n;

	float3 recv_pos = r_args.d_recv_focus_pos[recvIndex];
	float3 recv_normal = r_args.d_recv_normal[recvIndex];
	float3 imgplane_normal = normalize(h_args.d_helio_pos[helioIndex] - recv_pos);
	float cos_phi = dot(recv_normal, imgplane_normal);
	if (cos_phi < Epsilon) return;

	float3 reverse_dir = imgplane_normal;		// The normal of image plane
	float3* recv_v = r_args.d_recv_vertexes + 4 * recvIndex;
	float4* imgplane_m = h_args.d_imgplane_world2local + 4 * helioIndex;
	float2 proj_v[4];
	float3 inter_v;
	for (int i = 0; i < 4; ++i) {
		GeometryFunc::calcIntersection(reverse_dir, recv_pos, recv_v[i], reverse_dir, inter_v);
		inter_v = GeometryFunc::multMatrix(inter_v, imgplane_m);
		proj_v[i] = make_float2(inter_v.x, inter_v.z);
	}

	float2 row_gap = (proj_v[3] - proj_v[0]) / m;
	float2 col_gap = (proj_v[1] - proj_v[0]) / n;

	float l_w_ratio = h_args.d_lw_ratio[helioIndex];
	float4 tmp_x = make_float4(
		(proj_v[0] + i*row_gap + j*col_gap).x,
		(proj_v[0] + (i + 1)*row_gap + j*col_gap).x,
		(proj_v[0] + (i + 1)*row_gap + (j + 1)*col_gap).x,
		(proj_v[0] + i*row_gap + (j + 1)*col_gap).x
	);

	float4 tmp_y = make_float4(
		(proj_v[0] + i*row_gap + j*col_gap).y,
		(proj_v[0] + (i + 1)*row_gap + j*col_gap).y,
		(proj_v[0] + (i + 1)*row_gap + (j + 1)*col_gap).y,
		(proj_v[0] + i*row_gap + (j + 1)*col_gap).y
	);

	float sum = gl.calcInte(tmp_x, tmp_y, h_args.sigma, l_w_ratio) * h_args.d_factor[helioIndex] * cos_phi;

	atomicAdd(d_total_energy, sum);
}