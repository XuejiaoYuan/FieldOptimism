#include "hip/hip_runtime.h"
#include "FluxIntegral.cuh"

__global__ void calcFieldFluxIntegral(IntegralHelioDeviceArgumet h_args, ReceiverDeviceArgument r_args, GaussLegendre gl, float* d_total_energy, const int m, const int n) {
	float res = calcFluxIntegralCore(h_args, r_args, gl, m, n);
	if (res < Epsilon) return;
	atomicAdd(d_total_energy, res);
}

__global__ void calcHelioFluxIntegral(IntegralHelioDeviceArgumet h_args, ReceiverDeviceArgument r_args, GaussLegendre gl, float* d_helio_energy, const int m, const int n) {
	int myId = GeometryFunc::getThreadId();
	if (myId >= m*n*h_args.numberOfHeliostats*r_args.numberOfReceivers) return;

	float res = calcFluxIntegralCore(h_args, r_args, gl, m, n);

	int helioIndex = myId / (m*n*r_args.numberOfReceivers);
	atomicAdd(d_helio_energy + helioIndex, res);
}

//__device__ float calcSigma(IntegralHelioDeviceArgumet& h_args, ReceiverDeviceArgument& r_args, int helioIndex, int recvIndex)
//{
//	float3 helio_pos = h_args.d_helio_pos[helioIndex];
//	float3 recv_pos = r_args.d_recv_focus_pos[recvIndex];
//	float dis = norm(helio_pos, recv_pos);
//
//	float3 helio_normal = h_args.d_helio_normals[helioIndex];
//	float3 recv_normal = r_args.d_recv_normal[recvIndex];
//	float3 reflect_dir = normalize(recv_pos - helio_pos);
//	float3 sunray_dir = -reflect(-reflect_dir, helio_normal);
//	float cos_w = abs(dot(sunray_dir, helio_normal));
//	float cos_rev = abs(dot(recv_normal, reflect_dir));
//	
//	float d = sqrt(h_args.d_helio_size.x * h_args.d_helio_size.y);
//	float Ht = d*(1 - cos_w);
//	float Ws = Ht;
//	float sigma_ast = Ht / (4 * dis);			// already simplified
//
//	float sigma_sun = SIGMA_SUN;
//	float sigma_s = SIGMA_S;
//	float sigma_bq = pow(2 * sigma_s, 2);
//	float sigma_t = 0;
//
//	float sigma_hf = sqrt(pow(dis, 2) * (pow(sigma_sun, 2) + pow(sigma_bq, 2) + pow(sigma_ast, 2) + pow(sigma_t, 2))) / sqrt(cos_rev);
//	return sigma_hf;
//}


__device__ float calcFluxIntegralCore(IntegralHelioDeviceArgumet& h_args, ReceiverDeviceArgument& r_args, GaussLegendre& gl, const int m, const int n) {
	int myId = GeometryFunc::getThreadId();
	if (myId >= m*n*h_args.numberOfHeliostats*r_args.numberOfReceivers) return -1;

	int helioIndex = myId / (m*n*r_args.numberOfReceivers);
	int recvIndex = (myId % (m*n*r_args.numberOfReceivers)) / (m*n);
	int row_col = (myId % (m*n*r_args.numberOfReceivers)) % (m*n);
	int i = row_col / n;
	int j = row_col % n;

	float3 recv_pos = r_args.d_recv_focus_pos[recvIndex];
	float3 recv_normal = r_args.d_recv_normal[recvIndex];
	float3 imgplane_normal = normalize(h_args.d_helio_pos[helioIndex] - recv_pos);
	float cos_phi = dot(recv_normal, imgplane_normal);
	if (cos_phi < Epsilon) return;

	float3 reverse_dir = imgplane_normal;		// The normal of image plane
	float3* recv_v = r_args.d_recv_vertexes + 4 * recvIndex;
	float4* imgplane_m = h_args.d_imgplane_world2local + 4 * helioIndex;
	float2 proj_v[4];
	float3 inter_v;
	for (int i = 0; i < 4; ++i) {
		GeometryFunc::calcIntersection(reverse_dir, recv_pos, recv_v[i], reverse_dir, inter_v);
		inter_v = GeometryFunc::multMatrix(inter_v, imgplane_m);
		proj_v[i] = make_float2(inter_v.x, inter_v.z);
	}

	float2 row_gap = (proj_v[3] - proj_v[0]) / m;
	float2 col_gap = (proj_v[1] - proj_v[0]) / n;

	float2 gauss_param = h_args.d_gauss_param[helioIndex];
	float l_w_ratio = gauss_param.x;
	float sigma = gauss_param.y;

	//float l_w_ratio = norm(proj_v[1], proj_v[0]) / norm(proj_v[3], proj_v[0]);

	float4 tmp_x = make_float4(
		(proj_v[0] + i*row_gap + j*col_gap).x,
		(proj_v[0] + (i + 1)*row_gap + j*col_gap).x,
		(proj_v[0] + (i + 1)*row_gap + (j + 1)*col_gap).x,
		(proj_v[0] + i*row_gap + (j + 1)*col_gap).x
	);

	float4 tmp_y = make_float4(
		(proj_v[0] + i*row_gap + j*col_gap).y,
		(proj_v[0] + (i + 1)*row_gap + j*col_gap).y,
		(proj_v[0] + (i + 1)*row_gap + (j + 1)*col_gap).y,
		(proj_v[0] + i*row_gap + (j + 1)*col_gap).y
	);

	//float sigma = calcSigma(h_args, r_args, helioIndex, recvIndex);

	float sum = gl.calcInte(tmp_x, tmp_y, sigma, l_w_ratio) * h_args.d_factor[helioIndex];// *cos_phi;

	return sum;
}