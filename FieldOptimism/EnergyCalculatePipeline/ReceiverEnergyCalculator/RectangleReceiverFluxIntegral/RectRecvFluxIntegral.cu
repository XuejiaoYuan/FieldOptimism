#include "hip/hip_runtime.h"
#include "RectRecvFluxIntegral.cuh"

void calcRectRecvEnergySum(int m, int n, int helioNum, IntegralHelioDeviceArgumet& h_args, ReceiverDeviceArgument& r_args, GaussLegendre& gl_handler, float* d_helio_energy){

	int nThreads = 512;
	dim3 nBlocks;
	GeometryFunc::setThreadsBlocks(nBlocks, nThreads, helioNum*m*n);

	calcHelioRectRecvFlux << <nBlocks, nThreads >> > (h_args, r_args, gl_handler, d_helio_energy, m, n);
	hipDeviceSynchronize();

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
	}
}


__global__ void calcRectRecvFluxSum(IntegralHelioDeviceArgumet h_args, ReceiverDeviceArgument r_args, GaussLegendre gl, float* d_total_energy, const int m, const int n) {
	float res = calcRectRecvFluxIntegralCore(h_args, r_args, gl, m, n);
	if (res < Epsilon) return;
	atomicAdd(d_total_energy, res);
}

__global__ void calcHelioRectRecvFlux(IntegralHelioDeviceArgumet h_args, ReceiverDeviceArgument r_args, GaussLegendre gl, float* d_helio_energy, const int m, const int n) {
	int myId = GeometryFunc::getThreadId();
	if (myId >= m*n*h_args.numberOfHeliostats*r_args.numberOfReceivers) return;

	float res = calcRectRecvFluxIntegralCore(h_args, r_args, gl, m, n);

	int helioIndex = myId / (m*n*r_args.numberOfReceivers);
	atomicAdd(d_helio_energy + helioIndex, res);
}


__device__ float calcRectRecvFluxIntegralCore(IntegralHelioDeviceArgumet& h_args, ReceiverDeviceArgument& r_args, GaussLegendre& gl, const int m, const int n) {
	int myId = GeometryFunc::getThreadId();
	if (myId >= m*n*h_args.numberOfHeliostats*r_args.numberOfReceivers) return -1;

	int helioIndex = myId / (m*n*r_args.numberOfReceivers);
	int recvIndex = (myId % (m*n*r_args.numberOfReceivers)) / (m*n);
	int row_col = (myId % (m*n*r_args.numberOfReceivers)) % (m*n);
	int i = row_col / n;
	int j = row_col % n;

	return calcRecvFluxIntegralCore(h_args, r_args, gl, helioIndex, recvIndex, i, j, m, n);
}

__device__ float calcRecvFluxIntegralCore(IntegralHelioDeviceArgumet& h_args, ReceiverDeviceArgument& r_args, GaussLegendre& gl, int helioIndex, int recvIndex, int i, int j, int m, int n) {
	int focus_idx = h_args.d_focus_index[helioIndex];
	float3 focus_pos = r_args.d_recv_focus_pos[focus_idx];
	float3 recv_normal = r_args.d_recv_normal[recvIndex];
	float3 imgplane_normal = normalize(h_args.d_helio_pos[helioIndex] - focus_pos);
	float cos_phi = dot(recv_normal, imgplane_normal);
	if (cos_phi < Epsilon) return;

	float3 reverse_dir = imgplane_normal;		// The normal of image plane
	float3* recv_v = r_args.d_recv_vertexes + 4 * recvIndex;
	float4* imgplane_m = h_args.d_imgplane_world2local + 4 * helioIndex;
	float2 proj_v[4];
	float3 inter_v;

	float3 h_center_bias = make_float3(0, 0, 0);
	float3 i_center_bias = make_float3(0, 0, 0);
	float rotate_theta = 0;
	if (h_args.d_center_bias) {
		h_center_bias = h_args.d_center_bias[helioIndex];
		GeometryFunc::calcIntersection(reverse_dir, focus_pos, h_center_bias, -reverse_dir, i_center_bias);
		i_center_bias = GeometryFunc::multMatrix(i_center_bias, imgplane_m);
		rotate_theta = h_args.d_rotate_theta[helioIndex];
	}

	for (int i = 0; i < 4; ++i) {
		GeometryFunc::calcIntersection(reverse_dir, focus_pos, recv_v[i], reverse_dir, inter_v);
		inter_v = GeometryFunc::multMatrix(inter_v, imgplane_m);
		proj_v[i] = make_float2(inter_v.x - i_center_bias.x, inter_v.z - i_center_bias.z);
		float2 trans_v;
		trans_v.x = proj_v[i].x*cos(rotate_theta) + proj_v[i].y*sin(rotate_theta);
		trans_v.y = proj_v[i].y*cos(rotate_theta) - proj_v[i].x*sin(rotate_theta);
		proj_v[i] = trans_v;
	}

	float2 row_gap = (proj_v[3] - proj_v[0]) / m;
	float2 col_gap = (proj_v[1] - proj_v[0]) / n;

	float2 gauss_param = h_args.d_gauss_param[helioIndex];
	float l_w_ratio = gauss_param.x;
	float sigma = gauss_param.y;

	float4 tmp_x = make_float4(
		(proj_v[0] + i*row_gap + j*col_gap).x,
		(proj_v[0] + (i + 1)*row_gap + j*col_gap).x,
		(proj_v[0] + (i + 1)*row_gap + (j + 1)*col_gap).x,
		(proj_v[0] + i*row_gap + (j + 1)*col_gap).x
	);

	float4 tmp_y = make_float4(
		(proj_v[0] + i*row_gap + j*col_gap).y,
		(proj_v[0] + (i + 1)*row_gap + j*col_gap).y,
		(proj_v[0] + (i + 1)*row_gap + (j + 1)*col_gap).y,
		(proj_v[0] + i*row_gap + (j + 1)*col_gap).y
	);

	float sum = gl.calcInte(tmp_x, tmp_y, sigma, l_w_ratio) * h_args.d_factor[helioIndex];

	return sum;

}