#include "hip/hip_runtime.h"
#include "CylinderRecvFluxIntegral.cuh"

void calcCylinderRecvEnergySum(int m, int n, int helioNum, IntegralHelioDeviceArgumet& h_args, ReceiverDeviceArgument& r_args, GaussLegendre& gl_handler, float* d_helio_energy)
{
	int nThreads = 512;
	dim3 nBlocks;
	GeometryFunc::setThreadsBlocks(nBlocks, nThreads, helioNum*m*n);

	calcHelioCylinderRecvFlux << <nBlocks, nThreads >> > (h_args, r_args, gl_handler, d_helio_energy, m, n);
	hipDeviceSynchronize();

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
	}
}

__global__ void calcCylinderRecvFluxSum(IntegralHelioDeviceArgumet h_args, ReceiverDeviceArgument r_args, GaussLegendre gl, float* d_total_energy, const int m, const int n) {
	float res = calcCylinderRecvFluxIntegralCore(h_args, r_args, gl, m, n);
	if (res < Epsilon) return;
	atomicAdd(d_total_energy, res);
}

__global__ void calcHelioCylinderRecvFlux(IntegralHelioDeviceArgumet h_args, ReceiverDeviceArgument r_args, GaussLegendre gl, float* d_helio_energy, const int m, const int n) {
	int myId = GeometryFunc::getThreadId();
	if (myId >= m*n*h_args.numberOfHeliostats) return;

	float res = calcCylinderRecvFluxIntegralCore(h_args, r_args, gl, m, n);

	int helioIndex = myId / (m*n*r_args.numberOfReceivers);
	atomicAdd(d_helio_energy + helioIndex, res);
}


__device__ float calcCylinderRecvFluxIntegralCore(IntegralHelioDeviceArgumet& h_args, ReceiverDeviceArgument& r_args, GaussLegendre& gl, const int m, const int n) {
	int myId = GeometryFunc::getThreadId();
	if (myId >= m*n*h_args.numberOfHeliostats) return -1;

	int helioIndex = myId / (m*n);
	int recvIndex = helioIndex;
	int row_col = myId % (m*n);
	int i = row_col / n;
	int j = row_col % n;

	return calcRecvFluxIntegralCore(h_args, r_args, gl, helioIndex, recvIndex, i, j, m, n);
}