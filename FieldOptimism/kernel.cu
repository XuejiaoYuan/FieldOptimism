#include "hip/hip_runtime.h"



#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <stdio.h>
#include <string>
#include <iostream>
#include "DataStructure\Timer.h"
using namespace std;




class A {
	int m;
public:
	A() :m(1) {}
	__device__ __host__ int getM() { return m; }
	void* operator new(size_t len) {
		void* ptr;
		hipMallocManaged(&ptr, len);
		return ptr;
	}
};

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void addWithCuda2(int* c, const int*a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	Timer t;
	t.resetStart();
    const int arraySize = 5;
    const int A[arraySize] = { 1, 2, 3, 4, 5 };
    const int B[arraySize] = { 10, 20, 30, 40, 50 };
	int C[arraySize] = { 0 };

	addWithCuda(C, A, B, arraySize);

	t.printDuration("hipMalloc");
	//if (cudaStatus != hipSuccess) {
	//    fprintf(stderr, "addWithCuda failed!");
	//    return 1;
	//}
	
	t.resetStart();
	int *a, *b;
	hipMallocManaged(&a, arraySize * sizeof(int));
	hipMallocManaged(&b, arraySize * sizeof(int));
	int *c = NULL;
	hipMallocManaged(&c, arraySize * sizeof(int));

	for (int i = 0; i < arraySize; ++i) {
		a[i] = i;
		b[i] = 10 * i;
	}
	for(int i=0; i<10000; ++i)
		addWithCuda2(c, a, b, arraySize);
	t.printDuration("hipMallocManaged");

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
	Timer t;
	t.resetStart();
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	t.printDuration("malloc memcpy");

	t.resetStart();
    // Launch a kernel on the GPU with one thread for each element.
	for (int i = 0; i < 10000; ++i) {
		addKernel << <1, size >> >(dev_c, dev_a, dev_b);
		cudaStatus = hipDeviceSynchronize();
	}
	t.printDuration("ѭ��");

    // Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


void addWithCuda2(int *c, const int*a, const int *b, unsigned int size) {
	//hipMallocManaged(&b, size * sizeof(int));
	//hipMallocManaged(&c, size * sizeof(int));

	addKernel << <1, size >> > (c, b, a);
	hipDeviceSynchronize();
}